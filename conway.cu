
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>

int rows = 1 << 13;
int cols = rows / 8;
int size = rows * cols;
int gens = 1000;
int space = size * sizeof(unsigned int);

void randomizeCells(unsigned int *cells) {
  for (int i = 0; i < size; ++i) {
    for (int j = 0; j < 4; ++j) {
      cells[i] <<= 8;
      cells[i] |= rand() & 0x11;
    }
  }
}

void printCells(unsigned int *cells) {
  std::cout << "\033[H\033[2J";
  for (int x = 0; x < rows && x < 16; ++x) {
    for (int y = 0; y < 16; ++y) {
      int idx = x * cols + y/8;
      bool on = ((cells[idx] >> (28-(y%8)*4))&1) == 1;
      std::cout << (on ? "o" : " ");
    }
    std::cout << std::endl;
  }
}

__global__
void neighborKernel(int size, int cols, unsigned int *cells, unsigned int *neighbors) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < size; i += stride) {
    neighbors[i] = 0;
    for (int dx = -1; dx <= 1; ++dx) {
      for (int dy = -1; dy <= 1; ++dy) {
        if (dx != 0 || dy != 0) {
          int ni = (size + i + dx * cols) % size;
          int ny = (size + ni + dy) % size;
          unsigned int alive = cells[ni];
          unsigned int last = cells[ny];

          switch (dy) {
          case 1:
            alive <<= 4;
            last >>= 28;
            alive |= last;
            break;
          case -1:
            alive >>= 4;
            last <<= 28;
            alive |= last;
            break;
          }

          neighbors[i] += alive;
        }
      }
    }
  }
}

__global__
void lifeKernel(int size, unsigned int *cells, unsigned int *neighbors) {
  unsigned int all_on = 0x11111111;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < size; i += stride) {
    unsigned int b4 = (neighbors[i] & (all_on << 2)) >> 2;
    unsigned int b2 = (neighbors[i] & (all_on << 1)) >> 1;
    unsigned int b1 = neighbors[i] & all_on;
    cells[i] = b2 & (b1 | cells[i]) & ~b4;
  }
}

int main(void) {
  unsigned int *gen1 = new unsigned int[size];
  randomizeCells(gen1);

  unsigned int *cells, *neighbors;

  hipMalloc(&cells, space);
  hipMalloc(&neighbors, space);
  unsigned int *result = new unsigned int[size];

  std::clock_t start, stop;
  start = std::clock();

  hipMemcpy(cells, gen1, space, hipMemcpyHostToDevice);
  for (int i = 0; i < gens; ++i) {
    neighborKernel<<<(size+255)/256, 256>>>(size, cols, cells, neighbors);
    lifeKernel<<<(size+255)/256, 256>>>(size, cells, neighbors);
  }
  hipMemcpy(result, cells, space, hipMemcpyDeviceToHost);

  stop = std::clock();
  float efficiency = float(long(rows) * rows * gens) / (stop - start) * CLOCKS_PER_SEC;

  hipFree(cells);
  hipFree(neighbors);

  std::cout << "C++ Efficiency in cellhz: " << efficiency << std::endl;
}
